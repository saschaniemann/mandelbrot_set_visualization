#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include<chrono>
#include<iostream>

__device__ void gradient_to_rgb(float gradient, unsigned char* r, unsigned char* g, unsigned char* b) {
    // hsv to rgb with h being the gradient and v and s fixed
    float h = gradient;
    float v = 0.8;
    float s = 1.0;
    int i = (int)(h * 6);
    float f = h * 6 - i;
    float p = v * (1 - s);
    float q = v * (1 - f * s);
    float t = v * (1 - (1 - f) * s);
    switch (i % 6) {
        case 0: *r = (unsigned char)(v * 255); *g = (unsigned char)(t * 255); *b = (unsigned char)(p * 255); break;
        case 1: *r = (unsigned char)(q * 255); *g = (unsigned char)(v * 255); *b = (unsigned char)(p * 255); break;
        case 2: *r = (unsigned char)(p * 255); *g = (unsigned char)(v * 255); *b = (unsigned char)(t * 255); break;
        case 3: *r = (unsigned char)(p * 255); *g = (unsigned char)(q * 255); *b = (unsigned char)(v * 255); break;
        case 4: *r = (unsigned char)(t * 255); *g = (unsigned char)(p * 255); *b = (unsigned char)(v * 255); break;
        case 5: *r = (unsigned char)(v * 255); *g = (unsigned char)(p * 255); *b = (unsigned char)(q * 255); break;
    }
}

__global__ void mandelbrot(uint32_t *pixels, int width, int height, float resolution, float offsetX, float offsetY, int numberOfIterations){
    float scale = height / resolution;
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    float real = (thread_id % width - (width / 2)) / -scale + offsetX;
    float img = (thread_id / width - (height / 2)) / -scale + offsetY;

    float x = real;
    float tx = x;
    float y = img;
    int n;
    for(n = 0; n < numberOfIterations; n++) {
        // (x, y) = (x^2−y^2−real, 2xy−img)
        tx = x;
        x = x*x - y*y - real;
        y = 2*tx*y - img;
        // if leaving circle with r=2, c=(0,0): break
        if(sqrt(x*x+y*y) > 2) {
            break;
        }
    }
    // in mandelbrot set
    if(n == numberOfIterations) {
        pixels[thread_id] = 0xff000000;
    }
    // not in mandelbrot set
    else {
        unsigned char r, g, b;
        float gradient = ((float) n) / numberOfIterations;
        gradient_to_rgb(gradient, &r, &g, &b);
        // convert rgb to one 32b uint
        pixels[thread_id] = r << 16 | g << 8 | b | 0xff000000;
    }
}

void call_kernel(uint32_t *pixels, int width, int height, float resolution, float offsetX, float offsetY){
    // Allocate memory.
    uint32_t *pixelsGPU; 
    int size = sizeof(uint32_t) * width * height;
    hipMalloc(&pixelsGPU, size);

    // copy from CPU to GPU
    hipMemcpy(pixelsGPU, pixels, size, hipMemcpyHostToDevice);

    // run mandelbrot kernel
    int threadsPerBlock = 256;
    int numberOfBlocks = ((width*height + threadsPerBlock -1) / threadsPerBlock);

    std::chrono::steady_clock::time_point beginKernel = std::chrono::steady_clock::now();
    mandelbrot<<<numberOfBlocks, threadsPerBlock>>>(pixelsGPU, width, height, resolution, offsetX, offsetY, 250);

    // wait for GPU to finish and copy from GPU to CPU
    hipDeviceSynchronize();
    std::chrono::steady_clock::time_point endKernel = std::chrono::steady_clock::now();
    std::cout << "Time for running kernel: " << std::chrono::duration_cast<std::chrono::milliseconds>(endKernel - beginKernel).count() << "[ms]" << std::endl;


    std::chrono::steady_clock::time_point beginCpyToHost = std::chrono::steady_clock::now();
    hipMemcpy(pixels, pixelsGPU, size, hipMemcpyDeviceToHost);
    std::chrono::steady_clock::time_point endCpyToHost = std::chrono::steady_clock::now();
    std::cout << "Time for copy to host: " << std::chrono::duration_cast<std::chrono::milliseconds>(endCpyToHost - beginCpyToHost).count() << "[ms]" << std::endl;

    hipFree(pixelsGPU);
}