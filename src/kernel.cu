#include "hip/hip_runtime.h"
#include "kernel.cuh"

__device__ void gradient_to_rgb(float gradient, unsigned char* r, unsigned char* g, unsigned char* b) {
    // hsv to rgb with h being the gradient and v and s fixed
    float h = gradient;
    float v = 0.8;
    float s = 1.0;
    int i = (int)(h * 6);
    float f = h * 6 - i;
    float p = v * (1 - s);
    float q = v * (1 - f * s);
    float t = v * (1 - (1 - f) * s);
    switch (i % 6) {
        case 0: *r = (unsigned char)(v * 255); *g = (unsigned char)(t * 255); *b = (unsigned char)(p * 255); break;
        case 1: *r = (unsigned char)(q * 255); *g = (unsigned char)(v * 255); *b = (unsigned char)(p * 255); break;
        case 2: *r = (unsigned char)(p * 255); *g = (unsigned char)(v * 255); *b = (unsigned char)(t * 255); break;
        case 3: *r = (unsigned char)(p * 255); *g = (unsigned char)(q * 255); *b = (unsigned char)(v * 255); break;
        case 4: *r = (unsigned char)(t * 255); *g = (unsigned char)(p * 255); *b = (unsigned char)(v * 255); break;
        case 5: *r = (unsigned char)(v * 255); *g = (unsigned char)(p * 255); *b = (unsigned char)(q * 255); break;
    }
}

__global__ void mandelbrot(uint32_t *pixels, int width, int height, float resolution, float offsetX, float offsetY, int numberOfIterations){
    float scale = height / resolution;
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    float real = (thread_id % width - (width / 2)) / -scale + offsetX;
    float img = (thread_id / width - (height / 2)) / -scale + offsetY;

    float x = real;
    float tx = x;
    float y = img;
    int n;
    for(n = 0; n < numberOfIterations; n++) {
        tx = x;
        x = x*x - y*y - real;
        y = 2*tx*y - img;
        // (x2−y2−a, 2xy−b)
        if(sqrt(x*x+y*y) > 2) {
            break;
        }
    }
    // not in mandelbrot set
    if(n == numberOfIterations) {
        pixels[thread_id] = 0x0;
    }
    else {
        unsigned char r, g, b;
        float gradient = ((float) n) / numberOfIterations;
        gradient_to_rgb(gradient, &r, &g, &b);

        pixels[thread_id] = r << 16 | g << 8 | b;
    }
}

void call_kernel(uint32_t *pixels, int width, int height, float resolution, float offsetX, float offsetY){
    // Declare Vektors on Host and Device
    uint32_t *pixelsGPU; 

    // Allocate memory.
    int size = sizeof(uint32_t) * width * height;
    
    hipMalloc(&pixelsGPU, size);

    // copy from CPU to GPU
    hipMemcpy(pixelsGPU, pixels, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 128;
    int numberOfBlocks = ((width*height + threadsPerBlock -1) / threadsPerBlock);
    std::cout << "blockSize: " << threadsPerBlock << ", numberOfBlocks: " << numberOfBlocks << std::endl;
    mandelbrot<<<numberOfBlocks, threadsPerBlock>>>(pixelsGPU, width, height, resolution, offsetX, offsetY, 250);
    // std::cout << pi << std::endl;

    hipDeviceSynchronize();
    // copy from GPU to CPU
    hipMemcpy(pixels, pixelsGPU, size, hipMemcpyDeviceToHost);
    std::cout << pixels[800*150 + 200] << std::endl;

    hipFree(pixelsGPU);
}